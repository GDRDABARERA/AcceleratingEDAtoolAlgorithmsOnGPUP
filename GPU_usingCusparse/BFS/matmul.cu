#include "hip/hip_runtime.h"
/* Program to do matrix multiplication in cuda
This program generates two matrices and multiply them*/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "helpers.cuh"

//Dimensions for matrix1
#define ROWS1 10
#define COLS1 20

//DImensions for matrix2
#define ROWS2 20
#define COLS2 15

/** CUDA kernel to do matrix multiplication**/
__global__ void matMul(int *matC_cuda, int *matA_cuda, int *matB_cuda){
	
	//derive the row and column based on thread configuration
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	
	//Limit calculations for valid indices
	if(row < ROWS1 && col < COLS2){
	
		int prod=0;
		int k;
		for(k=0;k<COLS1;k++){
			prod=prod+matA_cuda[row*COLS1+k]*matB_cuda[k*COLS2+col];
		}
		matC_cuda[row*COLS2+col]=prod;	
		
	}
	
}

int main(){
	
	//check whether dimensions are valid for matrix mutiplication
	if(COLS1!=ROWS2){
		printf("Matrix dimensions are invalid for matrix multiplication\n");
		exit(1);
	}
	
	//Initialize arrays in RAM
	int matA[ROWS1*COLS1];
	int matB[ROWS2*COLS2];
	int matC[ROWS1*COLS2];	
	
	//generate some values for matrixA
	int i,j;
	for(i=0;i<ROWS1;i++){
		for(j=0;j<COLS1;j++){
			matA[i*COLS1+j]=i+j;
		}
	}

	//print the matA
	printf("Matrix A : \n");
	for(i=0;i<ROWS1;i++){
		for(j=0;j<COLS1;j++){
			printf("%5d ",matA[i*COLS1+j]);
		}
		printf("\n");
	}		
	printf("\n");

	
	//generate values for matrixB
	for(i=0;i<ROWS2;i++){
		for(j=0;j<COLS2;j++){
			matB[i*COLS2+j]=i-j;
		}
	}

	//print the matB
	printf("Matrix B : \n");
	for(i=0;i<ROWS2;i++){
		for(j=0;j<COLS2;j++){
			printf("%5d ",matB[i*COLS2+j]);
		}
		printf("\n");
	}	
	printf("\n");

	/********************************** CUDA stuff starts here *******************************/
	
	//pointers for memory allocation in cudaa
	int *matA_cuda;
	int *matB_cuda;
	int *matC_cuda;
	
	//allocate memory in cuda
	checkCuda(hipMalloc((void **)&matA_cuda,sizeof(int)*ROWS1*COLS1));
	checkCuda(hipMalloc((void **)&matB_cuda,sizeof(int)*ROWS2*COLS2));
	checkCuda(hipMalloc((void **)&matC_cuda,sizeof(int)*ROWS1*COLS2));
	
	//copy memory from ram to cuda
	checkCuda(hipMemcpy(matA_cuda,matA,sizeof(int)*ROWS1*COLS1,hipMemcpyHostToDevice));
	checkCuda(hipMemcpy(matB_cuda,matB,sizeof(int)*ROWS2*COLS2,hipMemcpyHostToDevice));
	
	//multiply the matrices in cuda
	dim3 threadsPerBlock(16,16);
	dim3 numBlocks(ceil(COLS2/(float)16),ceil(ROWS1/(float)16));
	matMul<<<numBlocks,threadsPerBlock>>>(matC_cuda,matA_cuda,matB_cuda);
	checkCuda(hipGetLastError());
	
	//copy the answer back from cuda to ram
	checkCuda(hipMemcpy(matC,matC_cuda,sizeof(int)*ROWS1*COLS2,hipMemcpyDeviceToHost));

	//free the cuda memory
	checkCuda(hipFree(matA_cuda));
	checkCuda(hipFree(matB_cuda));
	checkCuda(hipFree(matC_cuda));
	
	/********************** CUDA stuff ends here ********************************/
	
	//print the answer
	printf("Answer : \n");	
	for(i=0;i<ROWS1;i++){
		for(j=0;j<COLS2;j++){
			printf("%5d ",matC[i*COLS2+j]);
		}
		printf("\n");
	}	

	return 0;

}
