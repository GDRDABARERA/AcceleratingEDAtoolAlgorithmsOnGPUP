#include "hip/hip_runtime.h"
  
#include <stdio.h>
#include <stdlib.h>
#include "helpers.cuh"

 struct Element{
        int value;
        int yPosition;
        
        struct Element * next;
};


// Inserts Node to the Linked List by Head Insertion - O(1)
// Returns address of head which is the newly created node.
struct Element * AddElement(struct Element * currentHead, int newValue, int newYposition){
    struct Element * newHead;
               //  = (struct Element *) malloc(sizeof(struct Element));
    hipMallocManaged(&newHead,sizeof(struct Element)*1);	
    checkCudaError();
  
    newHead->value = newValue;
    newHead->yPosition= newYposition;
    newHead->next = currentHead;
  
    return newHead;
}
 
 
__global__ void childtestkernelbfs( struct Element ** adjacencyList,
                        int * vertices,
                        int* inputV, int * finalLevel,int * output){
                        
                        
    
           
          int index = blockIdx.x*blockDim.x+threadIdx.x;
          
          
 
          
          
          if(*finalLevel>0 & index<*vertices){
                  printf("This is the FinalLevel : %d\n",*finalLevel);
                  
                  struct Element * traverse = adjacencyList[index];
                  
         
          
                if(traverse==NULL){
                        output[index]=0;
                                
                }else{
                       
                        int ans=0;
                        __syncthreads();
                        while (traverse != NULL) {
                               
                            int val = (traverse->value )*(inputV[traverse->yPosition]);
                            
                            
                            
                            ans= ans+ val;
                          printf("step in traversing %d* %d = val:%d , ans: %d\n",inputV[traverse->yPosition],traverse->value,val,ans);
                           // assert(traverse==NUL);
                           traverse = traverse->next;
                        } 
                       
                        
                        output[index]=ans;
                        
                         __syncthreads();
                }
         
                     /*     *finalLevel --;
                         printf("finalLevel is :%d", *finalLevel);  
	                 childtestkernelbfs<<<ceil(*vertices/256.0),256>>>(adjacencyList, vertices ,output, finalLevel, inputV);
	               
	                hipDeviceSynchronize(); */
	               // checkCudaError(); 
           
           
       
                  
        }
      

} 

__global__ void parentkernel(struct Element ** adjacencyList,
                        int * vertices,
                        int* inputV, int * finalLevel,int * output){
                        printf("parent kernel call \n");
                        childtestkernelbfs<<<1,*vertices>>>(adjacencyList, vertices , inputV, finalLevel,output);
                        hipDeviceSynchronize();
	               

}


int main(){


hipDeviceSetCacheConfig(hipFuncCachePreferL1);
//hipDeviceSetCacheConfig(hipFuncCachePreferShared:);

   //global variable asigning
   
   int * vertices;
   int * edges;
   int * inputArray;
   int * outputArray;
   int * startArrayCount;
   int * finalLevel;
   struct Element ** adjacencyList;
   
   
   //CPU variables only
   
   int v1,v2,i; //int levelSize=2;
   
   //unfied memory allocation for int values
   hipMallocManaged(&vertices,sizeof(int)*1);	
   checkCudaError();
   hipMallocManaged(&edges,sizeof(int)*1);	
   checkCudaError();
   hipMallocManaged(&startArrayCount,sizeof(int)*1);	
   checkCudaError();
   hipMallocManaged(&finalLevel,sizeof(int)*1);	
   checkCudaError();
   
   *finalLevel = 2;
   int modLevel= (*finalLevel)%2;
   //scan first line of graph data
    int noOfRows;
    FILE * graphFile =fopen("dataT/graphT.txt","r");
    fscanf(graphFile, "%d %d %d",&noOfRows, vertices, edges);
        printf("No fo rows %d, No of Cols %d, nnz %d \n",noOfRows,*vertices,*edges);    //- done
        

   hipMallocManaged(&(adjacencyList),sizeof(struct Elemnt*)*(*vertices));	
   checkCudaError();   

   //initialise main arrays
        for (i = 0; i < *vertices; ++i) {
                adjacencyList[i] = NULL;
            
         }
         
   //scan rest of the graph and create the adjacency list
   
        for (i = 0; i < *edges; ++i) {
                 int val;
                 fscanf(graphFile, "%d %d %d",&v1, &v2, &val);
          
                // Adding edge v1 --> v2
                adjacencyList[v1] = AddElement(adjacencyList[v1], v2,val);
  
         }
         
    // Printing Adjacency List
           printf("\nAdjacency List - of graph \n\n");
            for (i = 0; i < *vertices; ++i) {
                printf("adjacencyList[%d] -> ", i);
          
                struct Element * traverse = adjacencyList[i];
          
                while (traverse != NULL) {
                    printf("(%d,%d) -> ", traverse->value, traverse->yPosition);
                    traverse = traverse->next;
                }
          
                printf("NULL\n");
            }
            
     //scan the input vertices file
     
      FILE * vectorFile= fopen("dataT/input.txt","r");
      fscanf(vectorFile,"%d",startArrayCount);
      
    //unified memory allocation for input vertice vector
       hipMallocManaged(&inputArray,sizeof(int)*(*startArrayCount));	
       checkCudaError(); 
       hipMallocManaged(&outputArray,sizeof(int)*(*startArrayCount));	
       checkCudaError(); 
       
    //asign values for input vector
      for(i=0;i<*startArrayCount;i++){
                int tempVal; 
                fscanf(vectorFile,"%d",&tempVal);
                inputArray[i]= tempVal;
                printf("%d ,",inputArray[i]);
        }     

       
       // int count= *startArrayCount;
        
    //start Time measurement
	hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);	
	
	parentkernel<<<1,1>>>(adjacencyList, vertices , inputArray, finalLevel,outputArray);
	hipDeviceSynchronize();
	checkCudaError();

	//stop Time measurement
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stderr,"Time spent for kernel : %.10f seconds\n",elapsedtime/(float)1000);
	
	
	//print the output
	
	
     printf("\nthe final output vector is:\n");
        if(modLevel==0){
                for(i=0;i<*vertices;i++){
                        printf("%d,", outputArray[i]);
                
                }
        }else{
        
                 for(i=0;i<*vertices;i++){
                        printf("%d,", inputArray[i]);
                
                }
        
        }     
             
        return 0;
}






