
#include <hip/hip_runtime.h>
#include <stdio.h>
#define NUM_ELE 5

struct ListElem{

   int id;
   bool last;
   ListElem *next;
};

__global__ void test_kernel(ListElem *list){

  int count = 0;
  while (!(list->last)){
    printf("List element %d has id %d\n", count++, list->id);
    list = list->next;}
  printf("List element %d is the last item in the list\n", count);
}

int main(){
  ListElem *h_list, *my_list;
  hipHostAlloc(&h_list, sizeof(ListElem), hipHostMallocDefault);
  my_list = h_list;
  for (int i = 0; i < NUM_ELE-1; i++){
    my_list->id = i+101;
    my_list->last = false;
    hipHostAlloc(&(my_list->next), sizeof(ListElem), hipHostMallocDefault);
    my_list = my_list->next;}
  my_list->last = true;
  test_kernel<<<1,1>>>(h_list);
  hipDeviceSynchronize();
}
