#include "hip/hip_runtime.h"
/* ==========  ========== ========== ========= */
//         Breadth First Search (BFS)          //
//               Algorithm in CUDA         //
/* ========== ========== ========== ========== */
  
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "helpers.cuh"
#include <assert.h>
  

struct Edge {
    int vertex;
    struct Edge * next;
};

 struct Edge **  adjacencyList;
    // Size is made (vertices + 1) to use the
    // array as 1-indexed, for simplicity
    
 /*All global variables comes here**/   
    
    
  int * parent;                                    //int parent[vertices + 1];
    // Each element holds the Node value of its parent
  int  * level; int * startVertices;
  int * inputsCircuits;
  int * lev;
  int *flag;  
  int   startArrayCount;                                  //int level[vertices + 1];
    // Each element holds the Level value of that node 
//define variables in unified memory

  int  * vertices;
  int  * edges;
  int v1,v2,i;

 __device__ void recursiveTraverse(int lev, struct Edge * adjacencyList[],int level[],int parent[],int i,int vertices){
                int par;
         struct Edge * traverse;
            if ((level[i] == lev)&(i<vertices)) {
                flag = 1;
                traverse = adjacencyList[i];
                par = i;
                printf("%d \n",par);
                while (traverse != NULL) {
                    if (level[traverse->vertex] != -1) {
                        traverse = traverse->next;
                        continue;
                    }
  
                    level[traverse->vertex] = lev + 1;
                    printf("%d ",level[traverse->vertex]);
                    parent[traverse->vertex] = par;
                    traverse = traverse->next;
                    ++i;
                    recursiveTraverse(lev,adjacencyList,level,parent,i,vertices);
                    
                }
            }else if(i<vertices){
                 ++i;
                recursiveTraverse(lev,adjacencyList,level,parent,i,vertices);
            }
        
 }
  
// Inserts Node to the Linked List by Head Insertion - O(1)
// Returns address of head which is the newly created node.
struct Edge * AddEdge(struct Edge * currentHead, int newVertex)
{
    struct Edge * newHead
                 = (struct Edge *) malloc(sizeof(struct Edge));
  
    newHead->vertex = newVertex;
    newHead->next = currentHead;
  
    return newHead;
}
  
__global__ void BreadthFirstSearch(
                        struct Edge * adjacencyList[],
                        int vertices,
                        int parent[],
                        int level[],
                        int startVertices[],int count
                       ){
                       
                       
 int i;
    
    // 'lev' represents the level to be assigned
    // 'par' represents the parent to be assigned
    // 'flag' used to indicate if graph is exhausted
    
     hipStream_t s1;
     hipStreamCreateWithFlags(&s1,hipStreamNonBlocking);
  
   lev=0;
    for(i=0;i<count;i++){
         int k =startVertices[i];
         level[k] = lev;
    
    }
    // We start at startVertex
  
    while (flag) {
        flag = 0;
        recursiveTraverse<<<1,1,0,s1>>>(lev,adjacencyList,level,parent,0,vertices);
  
        ++lev;
    }
} 



 
int main()
{
        
        
        
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        hipSetDevice(0);
             

    
        int NNZ;
        int noOfRows,noOfCols;
        FILE * graphFile =fopen("graph.txt","r");
        fscanf(graphFile, "%d %d %d",&noOfRows, &noOfCols, &NNZ);
        printf("No fo rows %d, No of Cols %d, nnz %d \n",noOfRows,noOfCols,NNZ);    //- done
        vertices = noOfRows;
        edges =NNZ;

        

  hipMallocManaged(&parent,vertices*sizeof(int));
  hipMallocManaged(&level,vertices*sizeof(int));
  hipMallocManaged(&adjacencyList,vertices*sizeof(struct Edge *)); 
  hipMallocManaged(&vertice, 1*sizeof(int));
  hipMallocManaged(&edge, 1*sizeof(int));
  hipMallocManaged(&lev, 1*sizeof(int));
  hipMallocManaged(&flag, 1*sizeof(int));
  
  flag=1;
  
 // Must initialize your array
         for (i = 0; i < vertices; ++i) {
                adjacencyList[i] = NULL;
                parent[i] = 0;
                level[i] = -1;
         }
         
          for (i = 0; i < edges; ++i) {
                 int val;
                 fscanf(graphFile, "%d %d %d",&v1, &v2, &val);
          
                // Adding edge v1 --> v2
                adjacencyList[v1] = AddEdge(adjacencyList[v1], v2);
          
                // Adding edge v2 --> v1
                // Remove this if you want a Directed Graph
               // adjacencyList[v2] = AddEdge(adjacencyList[v2], v1);
         }
         
         // Printing Adjacency List
         printf("\nAdjacency List - of graph \n\n");
            for (i = 0; i < vertices; ++i) {
                printf("adjacencyList[%d] -> ", i);
          
                struct Edge * traverse = adjacencyList[i];
          
                while (traverse != NULL) {
                    printf("%d -> ", traverse->vertex);
                    traverse = traverse->next;
                }
          
                printf("NULL\n");
            }
            
        printf("geting starting list of inputs:\n");
            
        
         
         
        FILE * vectorFile= fopen("input.txt","r");
        fscanf(vectorFile,"%d",&startArrayCount);
        
     //  int inputsCircuits[startArrayCount];
     
        hipMallocManaged(&inputsCircuits,startArrayCount*sizeof(int));
        
        for(i=0;i<startArrayCount;i++){
                int tempVal; 
                fscanf(vectorFile,"%d",&tempVal);
                inputsCircuits[i]= tempVal;
                printf("%d ,",inputsCircuits[i]);
        }
        printf("\n");
        
        
        
        hipEvent_t start,stop;
	float elapsedtime;
	hipEventCreate(&start);
	hipEventRecord(start,0);	
  
          BreadthFirstSearch<<<1,1>>>(adjacencyList, vertices, parent, level, inputsCircuits ,startArrayCount);
     
          hipDeviceSynchronize();
	  checkCudaError();

	//stop Time measurement
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedtime,start,stop);
	fprintf(stderr,"Time spent for kernel : %.10f seconds\n",elapsedtime/(float)1000);
     
     
  
    // Printing Level and Parent Arrays
    printf("\nLevel and Parent Arrays -\n");
    for (i = 1; i <= vertices; ++i) {
        printf("Level of Vertex %d is %d, Parent is %d\n",
                                  i, level[i], parent[i]);
    }
  
    return 0;
}
